#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <stdio.h>
//���������� ����������, pochti gotovo

#define SIZE 1024

__global__ void addKernel(int *c, int *a, int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] * b[i];
}

int main()
{

	int *dev_a, *host_a, *dev_b, *host_b, *host_c, *dev_c;
	host_a = (int *)malloc(SIZE * sizeof(int));
	host_b = (int *)malloc(SIZE * sizeof(int));
	host_c = (int *)malloc(sizeof(int) * SIZE);
	host_c[0] = 0;
	for (int i = 0; i < SIZE; i++)
	{
		host_a[i] = i;
		host_b[i] = i;
		host_c[i] = 0;
	}
	StartTimer();
	hipMalloc((void**)&dev_a, sizeof(int) * SIZE);
	hipMalloc((void**)&dev_b, sizeof(int) * SIZE);
	hipMalloc((void**)&dev_c, sizeof(int) * SIZE);

	hipMemcpy(dev_a, host_a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, host_c, sizeof(int) * SIZE, hipMemcpyHostToDevice);

	addKernel<<<1, SIZE >>>(dev_c, dev_a, dev_b);

	hipDeviceSynchronize();

	hipMemcpy(host_a, dev_a, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(host_b, dev_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(host_c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_c);
	hipFree(dev_b);
	for (int i = 1; i < SIZE; i++ )
		host_c[0] += host_c[i];
	printf("%d", host_c[0]);
	double runtime = GetTimer();

	printf(" total: %f s\n", runtime / 1000);
	getchar();
	return 0;
}